
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
	printf("Buzingga! This function is running on GPU\n");
}

int main(void) {
	printf("My first CUDA code which will run on GPU\n");
	helloFromGPU <<<1, 10>>>();
	hipDeviceReset();
	return(0);
}
