
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) {
	printf("Buzingga! This function is running on GPU\n");
}

int main(void) {
	printf("My first CUDA code which will run on GPU\n");
	helloFromGPU <<<1, 10>>>();
	hipDeviceReset();
	//1. comment the above function and see what happens
	
	//2. replace cudaDeviceReset() with cudaDeviceSynchronize() and see what happens
	// cudaDeviceSynchronize();

	return(0);
}
